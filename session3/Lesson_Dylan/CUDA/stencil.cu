#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pch.h"


__global__ void jacobi()
{

}

__host__ void Jacobi_naiveGPU(const float *A, const float *b, float *x, const int ny, const int nx, const float threshold)
{
  float *d_A, *d_b, *d_x;

  // Allocate device matrices on GPU using hipMalloc
  hipMalloc(&d_A, ny*nx*sizeof(float));
  hipMalloc(&d_b, *q*sizeof(float));
  hipMalloc(&d_x, m*q*sizeof(float));
  cudaCheckErrors("hipMalloc failure");


  // Free the device matrices
  hipFree(d_A);
  hipFree(d_b);
  hipFree(d_x);
  cudaCheckErrors("hipFree failure");

}
