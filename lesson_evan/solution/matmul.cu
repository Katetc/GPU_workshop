#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pch.h"
#include <stdio.h>

__global__ void SharedMatmul(const float *a, const float *b, float *c, const int m, const int p, const int q) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocate shared memory
  __shared__ float s_a[1024];
  __shared__ float s_b[1024];

  // Declare a temporary variable to accumulate calculated values
  // in the C matrix
  float tmp = 0.0;

  // Sweep tile across matrix
  for (int i = 0; i < p; i += blockDim.x) {
    // Load in elements for this tile
    int shared_index = threadIdx.y * blockDim.x + threadIdx.x;
    s_a[shared_index] = a[row * p + i + threadIdx.x];
    s_b[shared_index] = b[i * q + threadIdx.y * q + col];

    // Wait for tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write resulting calculation to the c matrix
  c[row * q + col] = tmp;
}

__host__ void gpuMatmul(const float *h_A, const float *h_B, float *h_C, const int m, const int p, const int q)
{
  float *d_A, *d_B, *d_C;

  // Allocate device matrices
  hipMalloc(&d_A, m*p*sizeof(float));
  hipMalloc(&d_B, p*q*sizeof(float));
  hipMalloc(&d_C, m*q*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, m*p*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, p*q*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");

  // Threads per CTA dimension
  int thread_dim = 32;
  // Blocks per grid dimension (assumes thread_dim divides M and N evenly)
  int blocks_x = q / thread_dim;
  int blocks_y = m / thread_dim;
  // Use dim3 structs for block and grid dimensions
  dim3 threads(thread_dim, thread_dim);
  dim3 blocks(blocks_x, blocks_y);

  // Calcuate AxB=C on the device
  SharedMatmul<<<blocks, threads>>>(d_A, d_B, d_C, m, p, q);
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, m*q*sizeof(float), hipMemcpyDeviceToHost);

  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  cudaCheckErrors("hipFree failure");
}
