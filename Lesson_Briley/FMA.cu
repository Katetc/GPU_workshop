#include "hip/hip_runtime.h"
//#include "Wkshp_head.h"
#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdio.h>
#include <hip/hip_runtime.h> 

__global__ void NaiveMult(float *d_A, float *d_B, float *d_C, const int ny, const int nx)
{
        int row = threadIdx.y+(blockIdx.y*blockDim.y);
        int col = threadIdx.x +(blockIdx.x*blockDim.x);
        float fSum = 0.0f;

        if (row<ny && col<nx) {
                for(int k=0; k<nx; k++)
                {
                        fSum += d_A[row*nx+k]*d_B[k*nx +col];
                }
                d_C[row*nx+col] = fSum;
        }

}
 
__host__ void gpuMult(float *h_A, float *h_B, float *h_C, const int ny, const int nx)
{ 
	float *d_A, *d_B, *d_C; 
	const int MSizeBytes = ny*nx*sizeof(float); 

	//Allocate memory on device 
	hipMalloc((void**)&d_A, MSizeBytes);		
	hipMalloc((void**)&d_B, MSizeBytes); 
	hipMalloc((void**)&d_C, MSizeBytes); 

	//Copy input data to device 
	hipMemcpy(d_A, h_A, MSizeBytes, hipMemcpyHostToDevice); 
	hipMemcpy(d_B, h_B, MSizeBytes, hipMemcpyHostToDevice);

	//Kernel Invoke Paramters (2D grid and blocks) 
	int dimx = 16; 
	int dimy = 16; 

	dim3 block(dimx, dimy); //Block of 256 threads 
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y); //grid dimensions 

	//Multiplication 
	NaiveMult<< <grid, block>> >(d_A,d_B,d_C,ny,nx);	
	
	hipDeviceSynchronize(); 
	
	//Copy Results back 
	hipMemcpy(h_C, d_C, MSizeBytes, hipMemcpyDeviceToHost);
 
	//Memory Release 
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree(d_B);
	hipDeviceReset(); 	

}


