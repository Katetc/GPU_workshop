#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pch.h"
#include <stdio.h>
 
//Add the keyword to make it a GPU function
 void add_matrices(float* m1, float* m2, float* sum, int ny, int nx){

    // Calculate the thread idx and idy index using threadIdx, blockDim, and blockIdx 
    //  Uncomment the following lines and calculate the thread's index w.r.t. the grid
    // int idx = ?;
    // int idy = ?;

    // Calculate the global linear address
    // int gla_index = ?;

    // Calculate the addition if within the matrix
    // if ( ? < ? && ? < ?) ?;

}

//Add the keyword to make it a host function
 void gpu_matrix_add(const float *h_A, const float *h_B, float *h_C,\
   const int ny, const int nx)
{
  float *d_A, *d_B, *d_C;

  // 3. Allocate memory on GPU for the matrices
  //  Uncomment the following 3 lines and replace question marks with correct variables 
  //  and size in bytes
  // hipMalloc(? , ?*?*sizeof(float));
  // hipMalloc(? , ?*?*sizeof(float));
  // hipMalloc(? , ?*?*sizeof(float));

  // Check for any errors
  cudaCheckErrors("hipMalloc failure");

  // 4. Copy the matrices A, B from host to device
  //  Uncomment the following 3 lines and fill in the correct destination, source, 
  //  and size in bytes
  // hipMemcpy(?, ?, ?, hipMemcpyHostToDevice);
  // hipMemcpy(?, ?, ?, hipMemcpyHostToDevice);
  // hipMemcpy(?, ?, ?, hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");

  // Kernel Invoke Paramters (3D grid and blocks but only using 2 dimensions) 
  //  Any dimension not assigned is automatically set to 1
  // Decide the dimensions of a block
  int dimx = 16; 
  int dimy = 16; 
  // Block of dimx*dimy threads
  // Uncomment the following line and fill in the block dimensions 
  // dim3 block(?, ?); 

  // Based on the number of threads in each block, calculate the minimum
  //  number of blocks in the x and y dimensions for any size matrix
  // Uncomment the following line and fill in the calculation for blocks in each dimension
  // dim3 grid( ?, ?); 

  printf("Kernel launch dimensions: \n");
  printf("\tGrid size  : {%d, %d, %d} blocks.\n",grid.x, grid.y, grid.z);
  printf("\tBlock size : {%d, %d, %d} threads.\n",block.x, block.y, block.z);
  
  // 5. Launch the kernel to perform A+B=C on GPU
  //  Uncomment the following line and fill in the gridDim, blockDim, and input variables
  // add_matrices<<<? , ?>>>(?, ?, ?, ?, ?);
  cudaCheckErrors("kernel launch failure");

  // Block the CPU until GPU finishes execution
  //  What happens when this line stays commented out? Uncomment for correct results
  // hipDeviceSynchronize();

  // 6. Copy the result matrix C from device to host
  //  Uncomment the following command and fill in the correct destination, source, size in bytes,
  //  and direction of copy
  // hipMemcpy(?, ?, ?, ?);

  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");

  // 7. Free the memory on GPU
  //  Uncomment the following 3 lines and put the correct variable to free device memory
  // hipFree(?);
  // hipFree(?);
  // hipFree(?);
  cudaCheckErrors("hipFree failure");

  // 8. Reset Device
  hipDeviceReset();
}
