#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pch.h"
#include <stdio.h>
 
__global__ void add_matrices(float* m1, float* m2, float* sum, int m, int n){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int index = idx*m+idy; 
    if (idx < m && idy < n) sum[index] = m1[index]+m2[index];
}

__host__ void gpu_matrix_add(const float *h_A, const float *h_B, float *h_C,\
   const int dx, const int dy)
{
  float *d_A, *d_B, *d_C;

  // Allocate device matrices
  hipMalloc(&d_A, dx*dy*sizeof(float));
  hipMalloc(&d_B, dx*dy*sizeof(float));
  hipMalloc(&d_C, dx*dy*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, dx*dy*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, dx*dy*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");

  //Kernel Invoke Paramters (2D grid and blocks) 
  int dimx = 16; 
  int dimy = 16; 

  dim3 block(dimx, dimy); //Block of 256 threads 
  dim3 grid((dx+block.x-1)/block.x, (dy+block.y-1)/block.y); //grid dimensions 

  // Calcuate A+B=C on the device
  add_matrices<<<grid, block>>>(d_A, d_B, d_C, dx, dy);
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, dx*dy*sizeof(float), hipMemcpyDeviceToHost);

  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  cudaCheckErrors("hipFree failure");
  hipDeviceReset();
}
