#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdio.h>
#include <hip/hip_runtime.h> 


__global__ void NaiveFMA(float *d_A, float *d_B, float *d_C,float *d_D, const int row, const int col)
{
        int row_idx = threadIdx.y+(blockIdx.y*blockDim.y);
        int col_idx = threadIdx.x +(blockIdx.x*blockDim.x);
        float fSum = 0.0f;

        if (row_idx<row && col_idx<col) {
                for(int k=0; k<col; k++)
                {
                        fSum += d_A[row_idx*col+k]*d_B[k*col +col_idx];
                }
                d_D[row_idx*col+col_idx] = fSum + d_C[row_idx*col+col_idx];
        }
}
 
__host__ void gpuFMA(float *h_A, float *h_B, float *h_C, float *h_D, const int row, const int col)
{ 
	float *d_A, *d_B, *d_C, *d_D; ; 
	const int MSizeBytes = row*col*sizeof(float); 

	//Allocate memory on device 
	hipMalloc((void**)&d_A, MSizeBytes);		
	hipMalloc((void**)&d_B, MSizeBytes); 
	hipMalloc((void**)&d_C, MSizeBytes); 
	hipMalloc((void**)&d_D, MSizeBytes);

	//Copy input data to device 
	hipMemcpy(d_A, h_A, MSizeBytes, hipMemcpyHostToDevice); 
	hipMemcpy(d_B, h_B, MSizeBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, MSizeBytes, hipMemcpyHostToDevice);

	//Kernel Invoke Paramters (2D grid and blocks) 
	int dimx = 32; 
	int dimy = 32; 

	dim3 block(dimx, dimy); //Block of 256 threads 
	dim3 grid((col+block.x-1)/block.x, (row+block.y-1)/block.y); //grid dimensions 

	//Multiplication 
	NaiveFMA<< <grid, block>> >(d_A,d_B,d_C,d_D,row,col);	
	
	hipDeviceSynchronize(); 
	
	//Copy Results back 
	hipMemcpy(h_D, d_D, MSizeBytes, hipMemcpyDeviceToHost);
 

	//Memory Release 
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree(d_C);
	hipFree(d_D);

}


